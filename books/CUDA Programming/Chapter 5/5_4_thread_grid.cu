#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void what_is_my_id_2d_A(
    unsigned int * const block_x,
    unsigned int * const block_y,
    unsigned int * const thread,
    unsigned int * const calc_thread,
    unsigned int * const x_thread,
    unsigned int * const y_thread,
    unsigned int * const grid_dimx,
    unsigned int * const block_dimx,
    unsigned int * const grid_dimy,
    unsigned int * const block_dimy) {
  const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
  const unsigned int thread_idx = ((gridDim.x * blockDim.x) * idy) + idx;

  block_x[thread_idx]     = blockIdx.x;   // 线程网格X维度上的线程块索引
  block_y[thread_idx]     = blockIdx.y;   // 线程网格Y维度上的线程块索引
  thread[thread_idx]      = threadIdx.x;  // 线程块X维度上的线程索引
  calc_thread[thread_idx] = thread_idx;
  x_thread[thread_idx]    = idx;
  y_thread[thread_idx]    = idy;
  grid_dimx[thread_idx]   = gridDim.x;    // 线程网格X维度上线程块的数量
  block_dimx[thread_idx]  = blockDim.x;   // 一个线程块X维度上线程数量
  grid_dimy[thread_idx]   = gridDim.y;    // 线程网格Y维度上线程块的数量
  block_dimy[thread_idx]  = blockDim.y;   // 一个线程块Y维度上线程数量
}

#define ARRAY_SIZE_X 32
#define ARRAY_SIZE_Y 16

#define ARRAY_SIZE_IN_BYTES (ARRAY_SIZE_X * ARRAY_SIZE_Y * sizeof(unsigned int))

// Declare statically six arrays of ARRAY_SIZE each
unsigned int cpu_block_x[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_y[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_calc_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_xthread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_ythread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];

int main(int argc, char const *argv[]) {
  // 线程块，条纹布局
  // ┏━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━┓
  // ┃ 线程0~15，块0    ┆ 线程16~31，块0   ┃
  // ┃ 线程32~47，块0   ┆ 线程48~63，块0   ┃
  // ┃ 线程64~79，块0   ┆ 线程80~95，块0   ┃
  // ┃ 线程96~111，块0  ┆ 线程112~127，块0 ┃
  // ┣━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━┫
  // ┃ 线程0~15，块1    ┆ 线程16~31，块1   ┃
  // ┃ 线程32~47，块1   ┆ 线程48~63，块1   ┃
  // ┃ 线程64~79，块1   ┆ 线程80~95，块1   ┃
  // ┃ 线程96~111，块1  ┆ 线程112~127，块1 ┃
  // ┣━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━┫
  // ┃ 线程0~15，块2    ┆ 线程16~31，块2   ┃
  // ┃ 线程32~47，块2   ┆ 线程48~63，块2   ┃
  // ┃ 线程64~79，块2   ┆ 线程80~95，块2   ┃
  // ┃ 线程96~111，块2  ┆ 线程112~127，块2 ┃
  // ┣━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━┫
  // ┃ 线程0~15，块4    ┆ 线程16~31，块4   ┃
  // ┃ 线程32~47，块4   ┆ 线程48~63，块4   ┃
  // ┃ 线程64~79，块4   ┆ 线程80~95，块4   ┃
  // ┃ 线程96~111，块4  ┆ 线程112~127，块4 ┃
  // ┗━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━━┛
  // Total thread count = 32 * 4 = 128
  const dim3 threads_rect(32, 4);
  const dim3 blocks_rect(1, 4);

  // 线程块，方块布局
  // ┏━━━━━━━━━━━━━━━━━┳━━━━━━━━━━━━━━━━━┓
  // ┃ 线程0~15，块0    ┃ 线程0~15，块1    ┃
  // ┃ 线程16~31，块0   ┃ 线程16~31，块1   ┃
  // ┃ 线程32~47，块0   ┃ 线程32~47，块1   ┃
  // ┃ 线程48~63，块0   ┃ 线程48~63，块1   ┃
  // ┃ 线程64~79，块0   ┃ 线程64~79，块1   ┃
  // ┃ 线程80~95，块0   ┃ 线程80~95，块1   ┃
  // ┃ 线程96~111，块0  ┃ 线程96~111，块1  ┃
  // ┃ 线程112~127，块0 ┃ 线程112~127，块1 ┃
  // ┣━━━━━━━━━━━━━━━━━╋━━━━━━━━━━━━━━━━━┫
  // ┃ 线程0~15，块2    ┃ 线程0~15，块4    ┃
  // ┃ 线程16~31，块2   ┃ 线程16~31，块4   ┃
  // ┃ 线程32~47，块2   ┃ 线程32~47，块4   ┃
  // ┃ 线程48~63，块2   ┃ 线程48~63，块4   ┃
  // ┃ 线程64~79，块2   ┃ 线程64~79，块4   ┃
  // ┃ 线程80~95，块2   ┃ 线程80~95，块4   ┃
  // ┃ 线程96~111，块2  ┃ 线程96~111，块4  ┃
  // ┃ 线程112~127，块2 ┃ 线程112~127，块4 ┃
  // ┗━━━━━━━━━━━━━━━━━┻━━━━━━━━━━━━━━━━━┛
  // Total thread count = 16 * 8 = 128
  const dim3 threads_square(16, 8);
  const dim3 blocks_square(2, 2);

  // Declare pointers for GPU based params
  unsigned int * gpu_block_x;
  unsigned int * gpu_block_y;
  unsigned int * gpu_thread;
  unsigned int * gpu_calc_thread;
  unsigned int * gpu_xthread;
  unsigned int * gpu_ythread;
  unsigned int * gpu_grid_dimx;
  unsigned int * gpu_block_dimx;
  unsigned int * gpu_grid_dimy;
  unsigned int * gpu_block_dimy;

  // Allocate four arrays on the GPU
  hipMalloc((void **)&gpu_block_x, ARRAY_SIZE_IN_BYTES);
  hipMalloc((void **)&gpu_block_y, ARRAY_SIZE_IN_BYTES);
  hipMalloc((void **)&gpu_thread, ARRAY_SIZE_IN_BYTES);
  hipMalloc((void **)&gpu_calc_thread, ARRAY_SIZE_IN_BYTES);
  hipMalloc((void **)&gpu_xthread, ARRAY_SIZE_IN_BYTES);
  hipMalloc((void **)&gpu_ythread, ARRAY_SIZE_IN_BYTES);
  hipMalloc((void **)&gpu_grid_dimx, ARRAY_SIZE_IN_BYTES);
  hipMalloc((void **)&gpu_block_dimx, ARRAY_SIZE_IN_BYTES);
  hipMalloc((void **)&gpu_grid_dimy, ARRAY_SIZE_IN_BYTES);
  hipMalloc((void **)&gpu_block_dimy, ARRAY_SIZE_IN_BYTES);

  for (int kernel = 0; kernel < 2; kernel++) {
    switch (kernel) {
      case 0: {
        what_is_my_id_2d_A<<<blocks_rect, threads_rect>>>(
          gpu_block_x, gpu_block_y, gpu_thread, gpu_calc_thread,
          gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_block_dimx,
          gpu_grid_dimy, gpu_block_dimy);
      } break;
      case 1: {
        what_is_my_id_2d_A<<<blocks_square, threads_square>>>(
          gpu_block_x, gpu_block_y, gpu_thread, gpu_calc_thread,
          gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_block_dimx,
          gpu_grid_dimy, gpu_block_dimy);
      } break;
      default: exit(1); break;
    }

    // Copy back gpu results to the CPU
    hipMemcpy(cpu_block_x, gpu_block_x, ARRAY_SIZE_IN_BYTES,
               hipMemcpyDeviceToHost);
    hipMemcpy(cpu_block_y, gpu_block_y, ARRAY_SIZE_IN_BYTES,
               hipMemcpyDeviceToHost);
    hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES,
               hipMemcpyDeviceToHost);
    hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTES,
               hipMemcpyDeviceToHost);
    hipMemcpy(cpu_xthread, gpu_xthread, ARRAY_SIZE_IN_BYTES,
               hipMemcpyDeviceToHost);
    hipMemcpy(cpu_ythread, gpu_ythread, ARRAY_SIZE_IN_BYTES,
               hipMemcpyDeviceToHost);
    hipMemcpy(cpu_grid_dimx, gpu_grid_dimx, ARRAY_SIZE_IN_BYTES,
               hipMemcpyDeviceToHost);
    hipMemcpy(cpu_block_dimx, gpu_block_dimx, ARRAY_SIZE_IN_BYTES,
               hipMemcpyDeviceToHost);
    hipMemcpy(cpu_grid_dimy, gpu_grid_dimy, ARRAY_SIZE_IN_BYTES,
               hipMemcpyDeviceToHost);
    hipMemcpy(cpu_block_dimy, gpu_block_dimy, ARRAY_SIZE_IN_BYTES,
               hipMemcpyDeviceToHost);
    printf("\nKernel %d\n", kernel);
    // Iterate through the arrays and print
    for (int y = 0; y < ARRAY_SIZE_Y; y++) {
      for (int x = 0; x < ARRAY_SIZE_X; x++) {
        printf("CT %3u BKX %1u BKY %1u TID %2u YTID %2u XTID %2u "
          "GDX %1u BDX %1u GDY %1u BDY %1u\n",
          cpu_calc_thread[y][x], cpu_block_x[y][x], cpu_block_y[y][x],
          cpu_thread[y][x], cpu_ythread[y][x], cpu_xthread[y][x],
          cpu_grid_dimx[y][x], cpu_block_dimx[y][x],
          cpu_grid_dimy[y][x], cpu_block_dimy[y][x]);
      }
    }
  }

  // Free the arrays on the GPU as now we're done with them
  hipFree(gpu_block_x);
  hipFree(gpu_block_y);
  hipFree(gpu_thread);
  hipFree(gpu_calc_thread);
  hipFree(gpu_xthread);
  hipFree(gpu_ythread);
  hipFree(gpu_grid_dimx);
  hipFree(gpu_block_dimx);
  hipFree(gpu_grid_dimy);
  hipFree(gpu_block_dimy);

  return 0;
}
