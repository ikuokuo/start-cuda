#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <string>

#define TIME_PRECISION 6
#include "common/time_cost.hpp"
#include "mat.hpp"

// Thread block size
#define BLOCK_SIZE 16

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col) {
  return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, float value) {
  A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {
  Matrix Asub;
  Asub.width    = BLOCK_SIZE;
  Asub.height   = BLOCK_SIZE;
  Asub.stride   = A.stride;
  Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                       + BLOCK_SIZE * col];
  return Asub;
}

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const MatrixC &A, const MatrixC &B, MatrixC *C) {
  TIME_BEG_FUNC2;
  // Load A and B to device memory
  Matrix d_A;
  d_A.width = d_A.stride = A.width; d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(float);
  hipMalloc(&d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
  Matrix d_B;
  d_B.width = d_B.stride = B.width; d_B.height = B.height;
  size = B.width * B.height * sizeof(float);
  hipMalloc(&d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

  // Allocate C in device memory
  Matrix d_C;
  d_C.width = d_C.stride = C->width; d_C.height = C->height;
  size = C->width * C->height * sizeof(float);
  hipMalloc(&d_C.elements, size);

  // Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
  TIME_BEG_FUNC("MatMulKernel");
  MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  TIME_END_FUNC("MatMulKernel");

  // Read C from device memory
  hipMemcpy(C->elements, d_C.elements, size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
  TIME_END_FUNC2;
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(const Matrix A, const Matrix B, Matrix C) {
  // Block row and column
  int blockRow = blockIdx.y;
  int blockCol = blockIdx.x;

  // Each thread block computes one sub-matrix Csub of C
  Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

  // Each thread computes one element of Csub
  // by accumulating results into Cvalue
  float Cvalue = 0;

  // Thread row and column within Csub
  int row = threadIdx.y;
  int col = threadIdx.x;

  // Loop over all the sub-matrices of A and B that are
  // required to compute Csub
  // Multiply each pair of sub-matrices together
  // and accumulate the results
  for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

    // Get sub-matrix Asub of A
    Matrix Asub = GetSubMatrix(A, blockRow, m);

    // Get sub-matrix Bsub of B
    Matrix Bsub = GetSubMatrix(B, m, blockCol);

    // Shared memory used to store Asub and Bsub respectively
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load Asub and Bsub from device memory to shared memory
    // Each thread loads one element of each sub-matrix
    As[row][col] = GetElement(Asub, row, col);
    Bs[row][col] = GetElement(Bsub, row, col);

    // Synchronize to make sure the sub-matrices are loaded
    // before starting the computation
    __syncthreads();

    // Multiply Asub and Bsub together
    for (int e = 0; e < BLOCK_SIZE; ++e)
        Cvalue += As[row][e] * Bs[e][col];

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write Csub to device memory
  // Each thread writes one element
  SetElement(Csub, row, col, Cvalue);
}

int main(int argc, char const *argv[]) {
  (void)argc;
  (void)argv;

  std::string dashes(60, '-');

  // Matrix on CPU

  int m_w = 1024, m_h = 1024;

  std::cout << "A" << dashes << std::endl;
  MatrixC A(m_w, m_h);
  A.Fill([](int r, int c) {
    if (r == 0) return c;
    if (c == 0) return r;
    return 0;
  }).Print(10, 10, MatrixC::PrintFormat(4, 0));

  std::cout << std::endl << "B" << dashes << std::endl;
  MatrixC B(A);
  B.Print(10, 10, MatrixC::PrintFormat(4, 0));

  // MatMul

  std::cout << std::endl << "C=AB" << dashes << std::endl;
  MatrixC C(m_w, m_h);
  MatMul(A, B, &C);
  C.Print(10, 10, MatrixC::PrintFormat(9, 0));

  return 0;
}
