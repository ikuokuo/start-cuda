#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <iterator>

#include "common/time_cost.hpp"

#define ARRAY_SIZE 256

__global__ void kernel_array_add(
    const int * const lhs,
    const int * const rhs,
    int * const sum,
    size_t n) {
  const unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  sum[i] = lhs[i] + rhs[i];
}

int main(int argc, char const *argv[]) {
  int lhs[ARRAY_SIZE];
  int rhs[ARRAY_SIZE];
  int sum[ARRAY_SIZE];

  // fill values
  std::fill(lhs, lhs+ARRAY_SIZE, 2);
  std::fill(rhs, rhs+ARRAY_SIZE, 3);

  {
    size_t bytes = sizeof(int) * ARRAY_SIZE;

    int *gpu_lhs;
    int *gpu_rhs;
    int *gpu_sum;

    hipMalloc((void**)&gpu_lhs, bytes);
    hipMalloc((void**)&gpu_rhs, bytes);
    hipMalloc((void**)&gpu_sum, bytes);

    // host to device
    hipMemcpy(gpu_lhs, lhs, bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_rhs, rhs, bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_sum, sum, bytes, hipMemcpyHostToDevice);

    TIME_BEG("kernel_array_add");
    kernel_array_add<<<ARRAY_SIZE/64, 64>>>(
        gpu_lhs, gpu_rhs, gpu_sum, ARRAY_SIZE);
    TIME_END("kernel_array_add");

    // device to host
    hipMemcpy(lhs, gpu_lhs, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(rhs, gpu_rhs, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(sum, gpu_sum, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_lhs);
    hipFree(gpu_rhs);
    hipFree(gpu_sum);
  }

  // print sum
  std::cout << "sum[" << ARRAY_SIZE << "]:" << std::endl;
  std::copy(sum, sum+ARRAY_SIZE, std::ostream_iterator<int>(std::cout, " "));
  std::cout << std::endl;

  return 0;
}
